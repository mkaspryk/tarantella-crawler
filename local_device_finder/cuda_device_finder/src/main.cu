//=======================================================================
// Name            : main.cu
// Author          : Marcin Grzegorz Kaspryk
// Version         : 1.0.0
// Copyright       : ASL
// Description     : CUDA config module
//=======================================================================

// includes, system
#include <stdio.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes header(CUDA errors catcher)
#include "cuda_error.hpp"

#define CUDA_DEVICE_NOT_FOUND -1

int set_device(){

	hipDeviceProp_t prop;
	int i,count,set;
	float compute_capability;
	TRY(hipGetDeviceCount(&count));

	if(count==0){
		return CUDA_DEVICE_NOT_FOUND;
	}

	set=0;
	compute_capability=prop.major+(prop.minor*0.1);

	printf("Number of devices: %d\n", count);
	for (i = 0; i < count; ++i) {
		printf("----------------Device specification------------------\n");
	    TRY(hipGetDeviceProperties(&prop, i));
	    printf("Device number: %d\n", i);
	    printf("Device name: %s\n", prop.name);
	    printf("Compute capability: %d.%d\n", prop.major, prop.minor);
	    printf("Multi processor count: %d\n", prop.multiProcessorCount);
	    printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
	    printf("Max threads dimension: (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
	    printf("Max grid size: (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
	    printf("---------------------------------------------------------\n");
	    if(compute_capability<(prop.major+(prop.minor*0.1))){
	    	compute_capability=prop.major+(prop.minor*0.1);
	    	set=i;
	    }
	}
	printf("Setted device: %d",set);
	return set;
}


int main(int argc, char **argv)
{
	return set_device();
}

