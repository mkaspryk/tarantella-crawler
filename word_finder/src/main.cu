//=======================================================================
// Name            : main.cu
// Author          : Marcin Grzegorz Kaspryk
// Version         : 1.0.0
// Copyright       : ASL
// Description     : CUDA module - main function
//=======================================================================

// includes, system
#include <stdio.h>
#include <stdlib.h>
#include <cstring>

// includes CUDA
#include <hip/hip_runtime.h>

// includes header(Word Finder CUDA module)
#include "word_finder.cuh"

// includes header(Word Finder CPU module)
#include "cpu_word_finder.hpp"

// includes header(files_handling)
#include "files_handling.hpp"

// includes header(CUDA errors catcher)
#include "cuda_error.hpp"

// export C interface (reads content from files)
extern "C"
void readingFiles(int argc, char**argv, int *flag,char **dic_words,char **web_words);

// export C interface (CPU word finder)
extern "C"
void cpu_finder(int *count, char **dic_words, char **web_words, int long_dic, int long_web);

#define LONGEST_WORD 45
#define NOT_ENOUGH_PARAMETERS -1

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
	int flag=0;
	if(argc<4){
		flag=NOT_ENOUGH_PARAMETERS;
		return flag;
	}

	// uses only by strtol
	char *p;
	int i,j;
	// sets the GPU or if -1 -> CPU
	int set_device = strtol(argv[1], &p, 10);
	int long_dic = strtol(argv[2], &p, 10);
	int long_web = strtol(argv[3], &p, 10);

	char **dic_words = (char**)malloc(long_dic*sizeof(char *));
	char **web_words = (char**)malloc(long_web*sizeof(char *));

	//================================
	// data to collect
	int *count;
	count = (int*)malloc(long_dic * sizeof(int));
	//================================

	for(i=0;i<long_dic;++i){
		dic_words[i] = (char*)malloc(LONGEST_WORD * sizeof(char));
		count[i] = 0;
	}

	for(i=0;i<long_web;++i){
		web_words[i] = (char*)malloc(LONGEST_WORD * sizeof(char));
	}

	readingFiles(argc,argv,&flag,dic_words,web_words);

	if(flag!=0){return flag;}

	if(set_device==-1){
		cpu_finder(count, dic_words, web_words, long_dic, long_web);
	}else{
		TRY(hipSetDevice(set_device));
		finder(&set_device, &flag, count, dic_words, web_words, long_dic, long_web, LONGEST_WORD);
	}

	for(i=0;i<long_dic;++i){

		printf("%s\n",dic_words[i]);
	}

	for(i=0;i<long_web;++i){

		printf("%s\n",web_words[i]);
	}

	for(i=0;i<long_dic;++i){

		printf("%d: %d\n",i,count[i]);
	}

	// frees the memory
	for (i = 0; i < long_dic; ++i) {
		free(dic_words[i]);
	}
	for (i = 0; i < long_web; ++i) {
		free(web_words[i]);
	}
	free(count);

	printf("flag = %d\n",flag);

	return flag;
}
