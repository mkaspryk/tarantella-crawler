//=======================================================================
// Name            : main.cu
// Author          : Marcin Grzegorz Kaspryk
// Version         : 1.0.0
// Copyright       : ASL
// Description     : Word Finder CUDA module
//=======================================================================

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>

// word finder module
#include "word_finder.cuh"

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
	return finder(argc,argv);
}
