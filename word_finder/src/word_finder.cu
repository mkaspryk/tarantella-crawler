#include "hip/hip_runtime.h"
//=======================================================================
// Name            : word_finder.cu
// Author          : Marcin Grzegorz Kaspryk
// Version         : 1.0.0
// Copyright       : ASL
// Description     : CUDA module - Word Finder
//=======================================================================

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes header(Word Finder CUDA module)
#include "word_finder.cuh"

// includes header(CUDA errors catcher)
#include "cuda_error.hpp"

__global__ void finderKernel(int *count, char *dic_words, char *web_words,int *long_dic, int *long_web, int *longest_word){

	int j,x,tid = blockIdx.x*blockDim.x+threadIdx.x;

	while(tid< *long_dic){
		for(j=0;j<*long_web;++j){
			x=0;
			while(dic_words[(tid*(*longest_word))+x]==web_words[(j*(*longest_word))+x]){
				if(dic_words[(tid*(*longest_word))+x]=='\0'&&web_words[(j*(*longest_word))+x]=='\0'){
					++count[tid];
					break;
				}
				++x;
			}
		}
		tid+=blockDim.x*gridDim.x;
	}
}

void finder(int *set_device, int *flag, int *count, char *dic_words, char *web_words, int long_dic, int long_web, int LONGEST_WORD){

	char *dic_words_d, *web_words_d;
	int *count_d, *long_dic_d, *long_web_d, *longest_word_d;

	TRY(hipMalloc((void**)&count_d,long_dic*sizeof(int)));
	TRY(hipMalloc((void**)&dic_words_d,long_dic*LONGEST_WORD*sizeof(char*)));
	TRY(hipMalloc((void**)&web_words_d,long_web*LONGEST_WORD*sizeof(char*)));
	TRY(hipMalloc((void**)&long_dic_d,sizeof(int)));
	TRY(hipMalloc((void**)&long_web_d,sizeof(int)));
	TRY(hipMalloc((void**)&longest_word_d,sizeof(int)));

	TRY(hipMemcpy(count_d,count,long_dic*sizeof(int),hipMemcpyHostToDevice));
	TRY(hipMemcpy(dic_words_d,&(dic_words[0]),long_dic*LONGEST_WORD*sizeof(char),hipMemcpyHostToDevice));
	TRY(hipMemcpy(web_words_d,&(web_words[0]),long_web*LONGEST_WORD*sizeof(char),hipMemcpyHostToDevice));

	TRY(hipMemcpy(long_dic_d,&long_dic,sizeof(int),hipMemcpyHostToDevice));
	TRY(hipMemcpy(long_web_d,&long_web,sizeof(int),hipMemcpyHostToDevice));
	TRY(hipMemcpy(longest_word_d,&LONGEST_WORD,sizeof(int),hipMemcpyHostToDevice));

	finderKernel<<< (long_dic+1023)/1024,1024 >>>(count_d, dic_words_d, web_words_d, long_dic_d, long_web_d, longest_word_d);

	TRY(hipDeviceSynchronize());

	TRY(hipMemcpy(count,count_d,long_dic*sizeof(int),hipMemcpyDeviceToHost));

	// frees the device memory
	hipFree(count_d);
    hipFree(dic_words_d);
    hipFree(web_words_d);
    hipFree(long_dic_d);
    hipFree(long_web_d);
    hipFree(longest_word_d);

	hipDeviceReset();

	return;
}
