//=======================================================================
// Name            : word_finder.cu
// Author          : Marcin Grzegorz Kaspryk
// Version         : 1.0.0
// Copyright       : ASL
// Description     : CUDA module - Word Finder
//=======================================================================

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes header(Word Finder CUDA module)
#include "word_finder.cuh"

void finder(int *set_device, int *flag, int *count, char **dic_words, char **web_words, int long_dic, int long_web, int LONGEST_WORD){

	char **dic_words_d,**web_words_d;

	hipMalloc((void**)&dic_words_d,long_dic*LONGEST_WORD*sizeof(char));
	hipMalloc((void**)&web_words_d,long_web*LONGEST_WORD*sizeof(char));

	//hipMemcpy();

	//hipMemcpy();

//	int i;
//	for(i=0;i<long_dic;++i){
//		//dic_words_d[i] = (char*)malloc(LONGEST_WORD * sizeof(char));
//	}
//
//	for(i=0;i<long_web;++i){
//		//web_words_d[i] = (char*)malloc(LONGEST_WORD * sizeof(char));
//	}


	return;
}



