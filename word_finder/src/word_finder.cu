#include "hip/hip_runtime.h"
//=======================================================================
// Name            : word_finder.cu
// Author          : Marcin Grzegorz Kaspryk
// Version         : 1.0.0
// Copyright       : ASL
// Description     : CUDA module - Word Finder
//=======================================================================

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes header(Word Finder CUDA module)
#include "word_finder.cuh"

// includes header(CUDA errors catcher)
#include "cuda_error.hpp"

__global__ void finderKernel(int *count, char *dic_words, char *web_words,int *long_dic, int *long_web){

	int tid = blockIdx.x*blockDim.x+threadIdx.x;

	while(tid< *long_dic){

		int i=0;
		while(i< *long_web){
			if(dic_words[tid]==web_words[i]){
				count[tid]=i;
			}
			i++;
		}
		tid+=blockDim.x*gridDim.x;
	}
}

void finder(int *set_device, int *flag, int *count, char **dic_words, char **web_words, int long_dic, int long_web, int LONGEST_WORD){


	char *dic_words_d, *web_words_d;
	int *count_d, *long_dic_d, *long_web_d;

	TRY(hipMalloc((void**)&count_d,long_dic*sizeof(int)));
	TRY(hipMalloc((void**)&dic_words_d,long_dic*LONGEST_WORD*sizeof(char*)));
	TRY(hipMalloc((void**)&web_words_d,long_web*LONGEST_WORD*sizeof(char*)));
	TRY(hipMalloc((void**)&long_dic_d,sizeof(int)));
	TRY(hipMalloc((void**)&long_web_d,sizeof(int)));

	TRY(hipMemcpy(count_d,count,long_dic*sizeof(int),hipMemcpyHostToDevice));
	TRY(hipMemcpy(dic_words_d,dic_words,long_dic*LONGEST_WORD*sizeof(char),hipMemcpyHostToDevice));
	TRY(hipMemcpy(web_words_d,web_words,long_web*LONGEST_WORD*sizeof(char),hipMemcpyHostToDevice));

	TRY(hipMemcpy(long_dic_d,&long_dic,sizeof(int),hipMemcpyHostToDevice));
	TRY(hipMemcpy(long_web_d,&long_web,sizeof(int),hipMemcpyHostToDevice));

	finderKernel<<< (long_dic+127)/128,128 >>>(count_d, dic_words_d, web_words_d, long_dic_d, long_web_d);

	TRY(hipDeviceSynchronize());

	TRY(hipMemcpy(count,count_d,long_dic*sizeof(int),hipMemcpyDeviceToHost));

	// frees the device memory
	hipFree(count_d);
    hipFree(dic_words_d);
    hipFree(web_words_d);
    hipFree(long_dic_d);
    hipFree(long_web_d);

	hipDeviceReset();

	return;
}
