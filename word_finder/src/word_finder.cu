//=======================================================================
// Name            : word_finder.cu
// Author          : Marcin Grzegorz Kaspryk
// Version         : 1.0.0
// Copyright       : ASL
// Description     : CUDA module - Word Finder
//=======================================================================

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes header(Word Finder CUDA module)
#include "word_finder.cuh"

void finder(int *flag,char **dic_words,char **web_words,int long_dic,int long_web){

	return;
}



