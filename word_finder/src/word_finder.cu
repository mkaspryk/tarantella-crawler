#include "hip/hip_runtime.h"
//=======================================================================
// Name            : word_finder.cu
// Author          : Marcin Grzegorz Kaspryk
// Version         : 1.0.0
// Copyright       : ASL
// Description     : CUDA module - Word Finder
//=======================================================================

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes header(Word Finder CUDA module)
#include "word_finder.cuh"

// includes header(CUDA errors catcher)
#include "cuda_error.hpp"

__global__ void finderKernel(int *count, char *dic_words, char *web_words,int long_dic, int long_web){

	int tid = blockIdx.x;
	if(tid<long_dic){
		count[tid]++;
	}
}

void finder(int *set_device, int *flag, int *count, char **dic_words, char **web_words, int long_dic, int long_web, int LONGEST_WORD){

	char *dic_words_d=0,*web_words_d=0;
	int *count_d;

	TRY(hipMalloc((void**)&dic_words_d,long_dic*LONGEST_WORD*sizeof(char)));
	TRY(hipMalloc((void**)&web_words_d,long_web*LONGEST_WORD*sizeof(char)));
	TRY(hipMalloc((void**)&count_d,long_dic*LONGEST_WORD*sizeof(int)));

	TRY(hipMemcpy(dic_words_d,dic_words,long_dic*LONGEST_WORD*sizeof(char),hipMemcpyHostToDevice));
	TRY(hipMemcpy(web_words_d,web_words,long_web*LONGEST_WORD*sizeof(char),hipMemcpyHostToDevice));
	TRY(hipMemcpy(count_d,count,long_dic*LONGEST_WORD*sizeof(int),hipMemcpyHostToDevice));

	dim3  grid(1, 1, 1);
	dim3  threads(32, 1, 1);

	finderKernel<<< grid,threads >>>(count_d, dic_words_d, web_words_d, long_dic, long_web);

	TRY(hipDeviceSynchronize());

	TRY(hipMemcpy(count,count_d,long_dic*LONGEST_WORD*sizeof(int),hipMemcpyDeviceToHost));

	// frees the device memory
    hipFree(dic_words_d);
    hipFree(web_words_d);
    hipFree(count_d);

	hipDeviceReset();

	return;
}
